
#include <hip/hip_runtime.h>
#include<stdio.h>
#define CUDA_BLOCK_X 128
#define CUDA_BLOCK_Y 1
#define CUDA_BLOCK_Z 1

int ARRAY_SIZE(int *arr)
{
  return (sizeof(arr) / sizeof(arr[0]));
}

int main()
{
  int i_nom_1;
  int i;
  int vec1[3] = {(1), (2), (3)};
  int vec2[3] = {(4), (5), (6)};
  int *result;
  for (i = 1; i <= (ARRAY_SIZE(vec1) + 0) / 1; i += 1) {
    result[1 * i + -1] = 0;
  }
  for (i_nom_1 = 1; i_nom_1 <= (ARRAY_SIZE(vec1) + 0) / 1; i_nom_1 += 1) {
    result[1 * i_nom_1 + -1] = vec1[1 * i_nom_1 + -1] + vec2[1 * i_nom_1 + -1];
  }
  printf("%d \n",result[0]);
  return - 1;
}
